#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <sodium.h>
#include <thrust/transform_reduce.h>

#include <math.h>

#define N 100
#define M 100
#define UPPER_BOUND 40

using namespace std;

struct squar: public unary_function<int,int>

{
	__host__ __device__
	int operator()(const int &x) const{
		return x*x;
	}

};


int main(int argc, char**argv){

	thrust::device_vector <int> vec(N*M);
	
	unsigned long long int result=0;
	for(int i=0;i<N*M;i++){
		vec[i]=randombytes_uniform(UPPER_BOUND);
	}
	
	result=thrust::transform_reduce(vec.begin(),vec.end(),squar(),0,thrust::plus<int>());
	cout<<result<<" "<<sqrt(result);
	return 0;
}